#include "hip/hip_runtime.h"
#include "conv-transpose-1d.cuh"

static  __global__ void conv_transpose_1d_kernel(
        const int s0, const int p0, const int d0,
        const int kernel_size, const int input_size, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst) {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index >= output_size) {
        return;
    }
    //printf("idx: %d stride %d\n", global_index,s0);

    int out_index = global_index / dst_ne0;

    for (int c = 0; c < src0_ne2; c++)
    {

        int idx = global_index % dst_ne0;

        int kernel_offset = (src0_ne0 * src0_ne1 * out_index) + (c * src0_ne0);
        int input_offset = src1_ne0 * c;

        if(global_index == 3 && s0 == 2)
        {
        printf("idx: %d ???: %d\n", global_index,src0_ne2);

        printf("idx: %d kernel offset: %d\n", global_index,kernel_offset);
        printf("idx: %d input offset: %d\n", global_index,input_offset);
        }

        int upper_bound = idx > src1_ne0-1 ? src1_ne0-1 : (int)(idx/s0)*s0; //inclusive
        /*
        int upper_bound = 0;
        while (upper_bound < idx){
            upper_bound +=1;
        }*/


        int lower_bound = idx - src0_ne0 + 1 >= 0 ? (int)(idx/s0)*s0 - src0_ne0 + 1 : 0;

        int initial_weight_idx = idx > src0_ne0 -1 ? src0_ne0-1 : idx;

        if(global_index == 3 && s0 == 2)
        {
        printf("idx: %d initial_weight_idx: %d\n", global_index,initial_weight_idx);
        printf("idx: %d upper bound: %d\n", global_index, upper_bound);
        printf("idx: %d lower bound: %d\n", global_index, lower_bound);
        }

        for (int i = 0; i < src1_ne0; i++)
        {
            if (!(idx >= i*s0 && idx < i*s0 + src0_ne0))
            {
                continue;
            }
            int weight_idx = idx - i*s0;


            if(global_index == 3 && s0 == 2)
            {
            //printf("idx: %d partial sum: %d x %d \n", global_index,src0[kernel_offset + (initial_weight_idx-(i-lower_bound))] , src1[input_offset+i]);
            //printf("idx: %d kernel_index: %d\n", global_index, kernel_offset + (initial_weight_idx-(i-lower_bound)));
            //printf("idx: %d input_index: %d\n", global_index, initial_weight_idx-(i-lower_bound));

            //printf("idx: %d input_index: %d\n", global_index, input_offset+i);

            }
            int test1 = src0[kernel_offset + weight_idx];
            int test2 =  src1[input_offset+i];
            if(global_index == 3 && s0 == 2)
            {
            //printf("idx: %d partial sum: %d x %d \n", global_index,src0[kernel_offset + (initial_weight_idx-(i-lower_bound))] , src1[input_offset+i]);
            //printf("idx: %d kernel_index: %d\n", global_index, kernel_offset + (initial_weight_idx-(i-lower_bound)));
            //printf("idx: %d input_index: %d\n", global_index, initial_weight_idx-(i-lower_bound));

            //printf("idx: %d input_index: %d\n", global_index, input_offset+i);
            printf("idx: %d test: %d x %d\n", global_index, test1, test2);

            }
            dst[global_index] += test1 * test2;
        }
        //dst[idx] = 7;
    }
}

static void conv_transpose_1d_f32_f32_cuda(
        const int s0, const int p0, const int d0,
        const int kernel_size, const int input_size, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst,
        hipStream_t stream) {

    const int num_blocks = (output_size + CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE - 1) / CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE;
    conv_transpose_1d_kernel<<<num_blocks,CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE, 0, stream>>>(s0,p0,d0,kernel_size, input_size, output_size,
    src0_ne0, src0_ne1,  src0_ne2, src0_ne3,
    src1_ne0, src1_ne1,  src1_ne2, src1_ne3,
    dst_ne0,  dst_ne1,   dst_ne2,  dst_ne3,
  src0,src1, dst);
}

void ggml_cuda_op_conv_transpose_1d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;

    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int32_t * opts = (const int32_t *)dst->op_params;

    const int s0 = dst->op_params[0];
    const int p0 = 0;//opts[3];
    const int d0 = 1;//opts[4];

    const int64_t kernel_size = ggml_nelements(src0);
    const int64_t input_size = ggml_nelements(src1);
    const int64_t output_size =  ggml_nelements(dst);


    conv_transpose_1d_f32_f32_cuda( s0,p0,d0,kernel_size, input_size, output_size, 
    src0->ne[0],src0->ne[1],src0->ne[2],src0->ne[3],
    src1->ne[0],src1->ne[1],src1->ne[2],src1->ne[3],
    dst->ne[0],dst->ne[1],dst->ne[2],dst->ne[3],
    src0_d, src1_d, dst_d, stream);
}
