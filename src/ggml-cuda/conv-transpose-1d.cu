#include "hip/hip_runtime.h"
#include "conv-transpose-1d.cuh"

static  __global__ void conv_transpose_1d_kernel(
        const int s0, const int p0, const int d0,
        const int kernel_size, const int input_size, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst) {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index >= output_size) {
        return;
    }

    int out_index = global_index / dst_ne0;

    for (int c = 0; c < src0_ne2; c++)
    {

        int idx = global_index % dst_ne0;

        int kernel_offset = (src0_ne0 * src0_ne1 * out_index) + (c * src0_ne0);
        int input_offset = src1_ne0 * c;

        if (global_index == 5)
        {
        printf("idx: %d ???: %d\n", global_index,src0_ne2);

        printf("idx: %d kernel offset: %d\n", global_index,kernel_offset);
        printf("idx: %d input offset: %d\n", global_index,input_offset);
        }

        int upper_bound = idx > src1_ne0-1 ? src1_ne0-1 : idx; //inclusive
        
        int lower_bound = idx - src0_ne0 + 1 >= 0 ? idx - src0_ne0 + 1 : 0;

        int initial_weight_idx = idx > src0_ne0 -1 ? src0_ne0-1 : idx;

        if(global_index == 5)
        {
        printf("idx: %d initial_weight_idx: %d\n", global_index,initial_weight_idx);
        printf("idx: %d upper bound: %d\n", global_index, upper_bound);
        printf("idx: %d lower bound: %d\n", global_index, lower_bound);
        }

        for (int i = lower_bound; i <= upper_bound; i++)
        {
            if(global_index == 5)
            {
            printf("idx: %d partial sum: %d\n", global_index,src0[kernel_offset + (initial_weight_idx-(i-lower_bound))] * src1[input_offset+i]);
            printf("idx: %d kernel_index: %d\n", global_index, kernel_offset + (initial_weight_idx-(i-lower_bound)));

            }
            dst[global_index] += src0[kernel_offset + (initial_weight_idx-(i-lower_bound))] * src1[input_offset+i];
        }
        //dst[idx] = 7;
    }
}

static void conv_transpose_1d_f32_f32_cuda(
        const int s0, const int p0, const int d0,
        const int kernel_size, const int input_size, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst,
        hipStream_t stream) {

    const int num_blocks = (output_size + CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE - 1) / CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE;
    conv_transpose_1d_kernel<<<num_blocks,CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE, 0, stream>>>(s0,p0,d0,kernel_size, input_size, output_size,
    src0_ne0, src0_ne1,  src0_ne2, src0_ne3,
    src1_ne0, src1_ne1,  src1_ne2, src1_ne3,
    dst_ne0,  dst_ne1,   dst_ne2,  dst_ne3,
  src0,src1, dst);
}

void ggml_cuda_op_conv_transpose_1d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;

    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int32_t * opts = (const int32_t *)dst->op_params;

    const int s0 = 1;//opts[2];
    const int p0 = 0;//opts[3];
    const int d0 = 1;//opts[4];

    const int64_t kernel_size = ggml_nelements(src0);
    const int64_t input_size = ggml_nelements(src1);
    const int64_t output_size =  ggml_nelements(dst);


    conv_transpose_1d_f32_f32_cuda( s0,p0,d0,kernel_size, input_size, output_size, 
    src0->ne[0],src0->ne[1],src0->ne[2],src0->ne[3],
    src1->ne[0],src1->ne[1],src1->ne[2],src1->ne[3],
    dst->ne[0],dst->ne[1],dst->ne[2],dst->ne[3],
    src0_d, src1_d, dst_d, stream);
}
